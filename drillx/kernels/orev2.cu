#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "orev2.h"
#include "utils.h"
#include "keccak.h"

__device__ uint32_t global_best_difficulty = 0;
__device__ unsigned long long int global_best_nonce = 0;
__device__ uint32_t device_best_difficulty; 
__device__ unsigned long long int device_best_nonce; 

__device__ size_t noise[NOISE_SIZE_BYTES / USIZE_BYTE_SIZE];

extern "C" void set_noise(const size_t *data)
{
    for (int device = 0; device < device_count; ++device)
    {
        hipSetDevice(device);
        hipMemcpyToSymbol(HIP_SYMBOL(noise), data, NOISE_SIZE_BYTES, 0, hipMemcpyHostToDevice);
    }
}

extern "C" void get_noise(size_t *host_data)
{
    for (int device = 0; device < device_count; ++device)
    {
        hipSetDevice(device);
        hipMemcpyFromSymbol(host_data, HIP_SYMBOL(noise), NOISE_SIZE_BYTES, 0, hipMemcpyDeviceToHost);
    }
}

extern "C" void drill_hash(uint8_t *challenge, uint8_t *out, uint64_t round)
{
    const uint64_t FIXED_NONCE_RANGE = 1000000000ULL; // 1 billion

    if (round == 0)
    {
        unsigned long long int zero = 0;
        uint32_t zero_difficulty = 0;

        hipMemcpyToSymbol(HIP_SYMBOL(global_best_nonce), &zero, sizeof(zero), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(global_best_difficulty), &zero_difficulty, sizeof(zero_difficulty), 0, hipMemcpyHostToDevice);
    }

 // Allocate device memory for input data and results
uint8_t *d_challenge[device_count];
uint64_t *d_best_nonce[device_count];
uint32_t *d_best_difficulty[device_count];
for (int device = 0; device < device_count; ++device)
{
    hipSetDevice(device);
    hipError_t cuda_status;

    cuda_status = hipMalloc((void **)&d_challenge[device], 32);
    if (cuda_status != hipSuccess) {
        printf("hipMalloc failed for d_challenge on GPU %d: %s\n", device, hipGetErrorString(cuda_status));
        // Handle error
        
    }

    cuda_status = hipMemcpy(d_challenge[device], challenge, 32, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        printf("hipMemcpy failed for d_challenge on GPU %d: %s\n", device, hipGetErrorString(cuda_status));
        // Handle error
        
    }

    cuda_status = hipMalloc((void **)&d_best_nonce[device], sizeof(uint64_t));
    if (cuda_status != hipSuccess) {
        printf("hipMalloc failed for d_best_nonce on GPU %d: %s\n", device, hipGetErrorString(cuda_status));
        // Handle error
        
    }

    cuda_status = hipMalloc((void **)&d_best_difficulty[device], sizeof(uint32_t));
    if (cuda_status != hipSuccess) {
        printf("hipMalloc failed for d_best_difficulty on GPU %d: %s\n", device, hipGetErrorString(cuda_status));
        // Handle error
        
    }
}

uint64_t total_stride = number_blocks * number_threads;
uint64_t nonce_per_device = FIXED_NONCE_RANGE / device_count;

for (int device = 0; device < device_count; ++device)
{
    hipSetDevice(device);
    uint64_t start_nonce = device * nonce_per_device;
    printf("Launching kernel on GPU %d with start nonce %llu\n", device, (unsigned long long int)start_nonce);

    // Print all arguments before launching the kernel
    printf("Kernel arguments for GPU %d:\n", device);
    printf("d_challenge: %p\n", d_challenge[device]);
    printf("total_stride: %llu\n", total_stride);
    printf("round: %llu\n", round);
    printf("batch_size: %u\n", batch_size);
    printf("start_nonce: %llu\n", (unsigned long long int)start_nonce);
    printf("d_best_nonce: %p\n", d_best_nonce[device]);
    printf("d_best_difficulty: %p\n", d_best_difficulty[device]);

    hipError_t cuda_status;
    // Launch the kernel with error checking
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        printf("Previous CUDA error before kernel launch on GPU %d: %s\n", device, hipGetErrorString(cuda_status));
        // Handle error
        
    }

    kernel_start_drill<<<number_blocks / device_count, number_threads>>>(d_challenge[device], total_stride, round, batch_size, start_nonce, d_best_nonce[device], d_best_difficulty[device]);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        printf("CUDA error during kernel launch on GPU %d: %s\n", device, hipGetErrorString(cuda_status));
        // Handle error
        
    }
}

    for (int device = 0; device < device_count; ++device)
    {
        hipSetDevice(device);
        hipDeviceSynchronize();
    }

    unsigned long long int best_nonce = 0;
    uint32_t best_difficulty = 0;
    int best_device = -1;
    for (int device = 0; device < device_count; ++device)
    {
        unsigned long long int device_nonce;
        uint32_t device_difficulty;
        hipSetDevice(device);
        hipMemcpy(&device_nonce, d_best_nonce[device], sizeof(device_nonce), hipMemcpyDeviceToHost);
        hipMemcpy(&device_difficulty, d_best_difficulty[device], sizeof(device_difficulty), hipMemcpyDeviceToHost);

        if (device_difficulty > best_difficulty)
        {
            best_difficulty = device_difficulty;
            best_nonce = device_nonce;
            best_device = device;
        }
    }

    uint64_t global_best_nonce_value;
    uint32_t global_best_difficulty_value;
    hipMemcpyFromSymbol(&global_best_nonce_value, HIP_SYMBOL(global_best_nonce), sizeof(global_best_nonce), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&global_best_difficulty_value, HIP_SYMBOL(global_best_difficulty), sizeof(global_best_difficulty), 0, hipMemcpyDeviceToHost);

    printf("Best Hash Found By Device %d\n", best_device);

    memcpy(out, &global_best_nonce_value, sizeof(global_best_nonce_value));

    for (int device = 0; device < device_count; ++device)
    {
        hipSetDevice(device);
        hipFree(d_challenge[device]);
        hipFree(d_best_nonce[device]);
        hipFree(d_best_difficulty[device]);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}


__global__ void kernel_start_drill(
    uint8_t *d_challenge,
    uint64_t stride,
    uint64_t round,
    uint32_t batch_size,
    uint64_t start_nonce,
    uint64_t *device_best_nonce,
    uint32_t *device_best_difficulty)
{
    printf("here\n");
    uint64_t iters = 0;
    uint64_t nonce = start_nonce + threadIdx.x + (blockIdx.x * blockDim.x);
    uint64_t local_best_nonce = nonce;
    uint32_t local_best_difficulty = 0;
    uint8_t result[32];

    
    while (iters < batch_size)
    {
        kernel_drill_hash(d_challenge, &nonce, result);
        uint32_t hash_difficulty = difficulty(result);
        if (hash_difficulty > local_best_difficulty)
        {
            local_best_difficulty = hash_difficulty;
            local_best_nonce = nonce;
        }
        nonce += stride; 
        iters += 1;
    }

    *device_best_difficulty = local_best_difficulty;
    *device_best_nonce = local_best_nonce;

    uint32_t prev_global_best_difficulty = atomicMax(&global_best_difficulty, local_best_difficulty);
    if (local_best_difficulty > prev_global_best_difficulty) {
        atomicExch(&global_best_nonce, local_best_nonce);
    }
}



extern "C" void single_drill_hash(uint8_t *challenge, uint64_t nonce, uint8_t *out)
{
    // Allocate device memory for input and output data
    uint8_t *d_challenge, *d_out;
    uint64_t *d_nonce;

    hipMalloc((void **)&d_challenge, 32);
    hipMalloc((void **)&d_out, 32);
    hipMalloc((void **)&d_nonce, 8);

    // Copy the host data to the device
    hipMemcpy(d_challenge, challenge, 32, hipMemcpyHostToDevice);
    hipMemcpy(d_nonce, &nonce, 8, hipMemcpyHostToDevice);

    // Launch the kernel to perform the hash operation
    single_drill_hash_routine<<<1, 1>>>(d_challenge, d_nonce, d_out);

    // Retrieve the results back to the host
    hipMemcpy(out, d_out, 32, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_challenge);
    hipFree(d_out);
    hipFree(d_nonce);
}

__global__ void single_drill_hash_routine(uint8_t *d_challenge, uint64_t *nonce, uint8_t *out)
{
    kernel_drill_hash(d_challenge, nonce, out);
}

__device__ void kernel_drill_hash(uint8_t *d_challenge, uint64_t *d_nonce, uint8_t *d_out)
{
    // Initialize the state preimage: [challenge, nonce]
    uint8_t state_preimage[40];
    memcpy(state_preimage, d_challenge, 32);
    memcpy(state_preimage + 32, d_nonce, 8);

    // Calculate initial state using keccak
    uint8_t state[32];
    keccak(state_preimage, 40, state);

    // Perform the drilling and hashing operations
    drill(state);
    keccak(state, 32, d_out);
}

__device__ void mine(
    uint8_t *d_challenge,
    uint64_t nonce,
    uint32_t *local_best_difficulty,
    uint64_t *local_best_nonce)
{
    // Drillhash
    uint8_t result[32];
    kernel_drill_hash(d_challenge, &nonce, result);

    // Compute difficulty
    uint32_t d = difficulty(result);
    if (d > *local_best_difficulty)
    {
        *local_best_nonce = nonce;
        *local_best_difficulty = d;
    }
}

__device__ void drill(uint8_t *state)
{
    // Initialize r
    uint64_t r = initialize_r(state);

    for (uint64_t i = 0; i < 4; i++)
    {
        // // Fetch noise
        uint64_t idxs[8];
        indices(state, idxs);
        for (uint64_t j = 0; j < 8; j++)
        {
            state[8 * i + j] ^= do_reads(state, idxs[j], r);
        }

        // Do ops
        indices(state, idxs);
        for (uint64_t j = 0; j < OPS; j++)
        {
            r ^= op(idxs[j % 8], r, j);
        }

        // hash state
        uint8_t state_preimage[40];
        memcpy(state_preimage, state, 32);
        memcpy(state_preimage + 32, &r, 8);
        keccak(state_preimage, 40, state);
    }
}

__device__ uint64_t initialize_r(const uint8_t *state)
{
    uint8_t rbytes[8] = {0, 0, 0, 0, 0, 0, 0, 0};
    uint8_t c = 0;

    for (int i = 0; i < 8; i++)
    {
        rbytes[i] = state[c % 32];
        c ^= rbytes[i];
    }

    uint64_t r = 0;
    for (int i = 0; i < 8; i++)
    {
        r |= (uint64_t)rbytes[i] << (8 * i);
    }

    return r;
}

__device__ void indices(uint8_t *state, uint64_t *indices)
{
    for (int i = 0; i < 8; i++)
    {
        indices[i] = (uint64_t)state[4 * i] |
                     (uint64_t)(state[4 * i + 1]) << 8 |
                     (uint64_t)(state[4 * i + 2]) << 16 |
                     (uint64_t)(state[4 * i + 3]) << 24;
    }
}

__device__ uint8_t do_reads(uint8_t *state, uint64_t index, uint64_t r)
{
    for (int i = 0; i < READS; i++)
    {
        index ^= noise[index % NOISE_LEN] * (uint64_t)state[i % 32];
    }

    return (uint8_t)(noise[index % NOISE_LEN] >> (r % 8));
}

__device__ uint64_t op(uint64_t a, uint64_t b, uint64_t opcount)
{
    Opcode opcode = static_cast<Opcode>((opcount ^ b) % CARDINALITY);
    switch (opcode)
    {
    case Add:
        return a + b;
    case Sub:
        return a - b;
    case Mul:
        return a * b;
    case Div:
        if (a > b)
        {
            return a / saturating_add(b, 2);
        }
        else
        {
            return b / saturating_add(a, 2);
        }
    case Xor:
        return a ^ b;
    case Right:
        return a >> (b % 64);
    case Left:
        return a << (b % 64);
    }
}
